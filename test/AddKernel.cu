﻿#include "hip/hip_runtime.h"


#include <memory>
#include <stdexcept>
// #include <thrust/host_vector.h>
// #include <thrust/device_vector.h>

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

template <typename T> static auto createDeviceBuffer(std::size_t Size)
{
    struct DeleterType
    {
        void operator()(void *Buffer) const noexcept
        {
            hipFree(Buffer);
        }
    };

    T *Result = nullptr;
    hipError_t cudaStatus = hipMalloc(reinterpret_cast<void **>(&Result), Size * sizeof(T));
    if (cudaStatus != hipSuccess)
        throw std::runtime_error("hipMalloc failed!");

    return std::unique_ptr<T, DeleterType>(Result);
}

// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
        throw std::runtime_error("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

    // Allocate GPU buffers for three vectors (two input, one output)    .
    auto DeviceBufferA = ::createDeviceBuffer<int>(size);
    auto DeviceBufferB = ::createDeviceBuffer<int>(size);
    auto DeviceBufferC = ::createDeviceBuffer<int>(size);

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(DeviceBufferA.get(), a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
        throw std::runtime_error("hipMemcpy failed!");

    cudaStatus = hipMemcpy(DeviceBufferB.get(), b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
        throw std::runtime_error("hipMemcpy failed!");

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(DeviceBufferC.get(), DeviceBufferA.get(), DeviceBufferB.get());

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
        throw std::runtime_error(std::string("addKernel launch failed: ") + hipGetErrorString(cudaStatus));

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
        throw std::runtime_error(std::string("hipDeviceSynchronize failed after launching addKernel: ") +
                                 hipGetErrorString(cudaStatus));

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, DeviceBufferC.get(), size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
        throw std::runtime_error("hipMemcpy failed!");
}

void myCudaDeviceReset()
{
    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    auto cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
        throw std::runtime_error("hipDeviceReset failed!");
}
